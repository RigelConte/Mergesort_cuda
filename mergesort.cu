
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdlib>
#include <ctime>

#define CUDA_CHECK_ERROR(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
    if (code != hipSuccess) 
    {
        std::cerr << "GPUassert: " << hipGetErrorString(code) << " " << file << " " << line << std::endl;
        if (abort) exit(code);
    }
}

__global__ void merge(int *arr, int left, int mid, int right) {
    int i, j, k;
    int n1 = mid - left + 1;
    int n2 = right - mid;

    // Create temporary arrays
    int *L, *R;
    L = (int*)malloc(n1 * sizeof(int));
    R = (int*)malloc(n2 * sizeof(int));

    // Copy data to temporary arrays L[] and R[]
    for (i = 0; i < n1; i++)
        L[i] = arr[left + i];
    for (j = 0; j < n2; j++)
        R[j] = arr[mid + 1 + j];

    // Merge the temporary arrays back into arr[left..right]
    i = 0;
    j = 0;
    k = left;
    while (i < n1 && j < n2) {
        if (L[i] <= R[j]) {
            arr[k] = L[i];
            i++;
        } else {
            arr[k] = R[j];
            j++;
        }
        k++;
    }

    // Copy the remaining elements of L[], if there are any
    while (i < n1) {
        arr[k] = L[i];
        i++;
        k++;
    }

    // Copy the remaining elements of R[], if there are any
    while (j < n2) {
        arr[k] = R[j];
        j++;
        k++;
    }

    // Free temporary arrays
    free(L);
    free(R);
}

__global__ void mergeSort(int *arr, int left, int right) {
    if (left < right) {
        int mid = left + (right - left) / 2;

        // Sort first and second halves
        mergeSort<<<1,1>>>(arr, left, mid);
        mergeSort<<<1,1>>>(arr, mid + 1, right);

        // Merge the sorted halves
        merge<<<1,1>>>(arr, left, mid, right);
    }
}

void mergeSortCUDA(int *arr, int size) {
    int *d_arr;
    CUDA_CHECK_ERROR(hipMalloc((void **)&d_arr, size * sizeof(int)));
    CUDA_CHECK_ERROR(hipMemcpy(d_arr, arr, size * sizeof(int), hipMemcpyHostToDevice));
    mergeSort<<<1,1>>>(d_arr, 0, size - 1);
    CUDA_CHECK_ERROR(hipMemcpy(arr, d_arr, size * sizeof(int), hipMemcpyDeviceToHost));
    CUDA_CHECK_ERROR(hipFree(d_arr));
}

void printArray(int *arr, int size) {
    for (int i = 0; i < size; i++)
        std::cout << arr[i] << " ";
    std::cout << std::endl;
}

int main() {
    const int size = 10;
    int arr[size];
    srand(time(NULL));
    for (int i = 0; i < size; i++)
        arr[i] = rand() % 100;

    std::cout << "Original array: ";
    printArray(arr, size);

    mergeSortCUDA(arr, size);

    std::cout << "Sorted array: ";
    printArray(arr, size);

    return 0;
}
